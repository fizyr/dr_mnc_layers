#include "hip/hip_runtime.h"
// --------------------------------------------------------
// Multitask Network Cascade
// Modified from caffe-fast-rcnn (https://github.com/rbgirshick/caffe-fast-rcnn)
// Copyright (c) 2016, Haozhi Qi
// Licensed under The MIT License [see LICENSE for details]
// --------------------------------------------------------


#include "fast_rcnn_layers.hpp"
#include <iostream>
namespace caffe {

template <typename Dtype>
__global__ void SmoothL1Forward(const int n, const Dtype* in, Dtype* out,
    Dtype sigma2) {
  // f(x) = 0.5 * (sigma * x)^2          if |x| < 1 / sigma / sigma
  //        |x| - 0.5 / sigma / sigma    otherwise
  CUDA_KERNEL_LOOP(index, n) {
    Dtype val = in[index];
    Dtype abs_val = abs(val);
    if (abs_val < 1.0 / sigma2) {
      out[index] = 0.5 * val * val * sigma2;
    } else {
      out[index] = abs_val - 0.5 / sigma2;
    }
  }
}

template <typename Dtype>
void SmoothL1LossLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  int count = bottom[0]->count();
  caffe_gpu_sub(
      count,
      bottom[0]->gpu_data(),
      bottom[1]->gpu_data(),
      diff_.mutable_gpu_data());    // d := b0 - b1
  if (has_weights_) {
    // apply "inside" weights
    caffe_gpu_mul(
        count,
        bottom[2]->gpu_data(),
        diff_.gpu_data(),
        diff_.mutable_gpu_data());  // d := w_in * (b0 - b1)
  }
  SmoothL1Forward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, diff_.gpu_data(), errors_.mutable_gpu_data(), sigma2_);
  CUDA_POST_KERNEL_CHECK;

  if (has_weights_) {
    // apply "outside" weights
    caffe_gpu_mul(
        count,
        bottom[3]->gpu_data(),
        errors_.gpu_data(),
        errors_.mutable_gpu_data());  // d := w_out * SmoothL1(w_in * (b0 - b1))
  }

  Dtype loss;
  caffe_gpu_dot(count, ones_.gpu_data(), errors_.gpu_data(), &loss);
  top[0]->mutable_cpu_data()[0] = loss / bottom[0]->num();
}

template <typename Dtype>
__global__ void SmoothL1Backward(const int n, const Dtype* in, Dtype* out,
    Dtype sigma2) {
  // f'(x) = sigma * sigma * x         if |x| < 1 / sigma / sigma
  //       = sign(x)                   otherwise
  CUDA_KERNEL_LOOP(index, n) {
    Dtype val = in[index];
    Dtype abs_val = abs(val);
    if (abs_val < 1.0 / sigma2) {
      out[index] = sigma2 * val;
    } else {
      out[index] = (Dtype(0) < val) - (val < Dtype(0));
    }
  }
}

template <typename Dtype>
void SmoothL1LossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  // after forwards, diff_ holds w_in * (b0 - b1)
  int count = diff_.count();
  SmoothL1Backward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, diff_.gpu_data(), diff_.mutable_gpu_data(), sigma2_);
  CUDA_POST_KERNEL_CHECK;
  for (int i = 0; i < 2; ++i) {
    if (propagate_down[i]) {
      const Dtype sign = (i == 0) ? 1 : -1;
      const Dtype alpha = sign * top[0]->cpu_diff()[0] / bottom[i]->num();
      caffe_gpu_axpby(
          count,                           // count
          alpha,                           // alpha
          diff_.gpu_data(),                // x
          Dtype(0),                        // beta
          bottom[i]->mutable_gpu_diff());  // y
      if (has_weights_) {
        // Scale by "inside" weight
        caffe_gpu_mul(
            count,
            bottom[2]->gpu_data(),
            bottom[i]->gpu_diff(),
            bottom[i]->mutable_gpu_diff());
        // Scale by "outside" weight
        caffe_gpu_mul(
            count,
            bottom[3]->gpu_data(),
            bottom[i]->gpu_diff(),
            bottom[i]->mutable_gpu_diff());
      }
    }
  }
  Dtype* bottom_data = bottom[0]->mutable_cpu_data();
  Dtype* bottom_data2 = bottom[1]->mutable_cpu_data();
  Dtype* diff_print = diff_.mutable_cpu_data();
  Dtype* bottom_diff = bottom[0]->mutable_cpu_diff();
  // we print the gradient for RPN loss
  /*
  if (bottom[0]->shape()[0] == 1) {
    LOG(INFO) << "SmoothL1Debug: " << bottom[0]->shape_string();
    LOG(INFO) << "SmoothL1Debug: " << bottom[0]->shape()[3] << " " << bottom[0]->shape()[2] << " " << bottom[0]->shape()[1];
    LOG(INFO) << top[0]->cpu_diff()[0] << " " << bottom[0]->num();
    
    for (int w = 0; w < bottom[0]->shape()[3]; w++) {
      for (int h = 0; h < bottom[0]->shape()[2]; h++) {
        for (int c = 0; c < bottom[0]->shape()[1]; c++) {
          Dtype data = bottom_diff[c*bottom[0]->shape()[2]*bottom[0]->shape()[3] + h * bottom[0]->shape()[3] + w];
          if (data < 1e-12 && data > -1 * 1e-12) continue;
          std::cout << "(" << c << "," << h << "," << w << "): " << data << std::endl;
        }
      }
    }
  }
  */
}

INSTANTIATE_LAYER_GPU_FUNCS(SmoothL1LossLayer);

}  // namespace caffe
